#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;
using namespace sf;

// CUDA kernel for updating the grid
__global__ void updateGridKernel(uint8_t* gridCurrent, uint8_t* gridNext,
                                 int gridWidth, int gridHeight)
{
    // ! This is 1D, so we need to unpack it back to (x, y) coordinates
    int l = blockIdx.x * blockDim.x + threadIdx.x; // x index of cell
    int y = l / gridWidth;
    int x = l % gridWidth;

    if (x >= gridWidth || y >= gridHeight)
        return; // Boundary check

    int neighbors = 0;
    // Count neighbors of the current cell
    for (int dx = -1; dx <= 1; dx++)
    {
        for (int dy = -1; dy <= 1; dy++)
        {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && nx < gridWidth && ny >= 0 && ny < gridHeight &&
                !(dx == 0 && dy == 0))
            {
                neighbors += gridCurrent[nx + ny * gridWidth];
            }
        }
    }

    // Conway's Game of Life rules
    if (gridCurrent[l])
    {
        gridNext[l] = (neighbors == 2 || neighbors == 3); // Cell remains alive
    }
    else
    {
        gridNext[l] = (neighbors == 3); // Cell becomes alive
    }
}

void normalMemSimulate(RenderWindow& window, int threadsPerBlock,
                       vector<vector<uint8_t>>& gridCurrent,
                       vector<vector<uint8_t>>& gridNext, int gridWidth,
                       int gridHeight, int cellSize, string memoryType)
{
    uint8_t *d_gridCurrent, *d_gridNext;
    int N = gridWidth * gridHeight;
    size_t size = N * sizeof(uint8_t);
    // * Allocate Memory on GPU
    if (memoryType == "NORMAL")
    {
        hipMalloc(&d_gridCurrent, size);
        hipMalloc(&d_gridNext, size);
    }
    else if (memoryType == "PINNED")
    {
        hipHostMalloc(&d_gridCurrent, size);
        hipHostMalloc(&d_gridNext, size);
    }
    else if (memoryType == "MANAGED")
    {
        hipMallocManaged(&d_gridCurrent, size);
        hipMallocManaged(&d_gridNext, size);
    }

    // * Flatten the vectors
    vector<uint8_t> flatGridCurrent;
    vector<uint8_t> flatGridNext;
    flatGridCurrent.reserve(gridWidth *
                            gridHeight); // Reserve memory for efficiency
    flatGridNext.reserve(gridWidth *
                         gridHeight); // Reserve memory for efficiency

    for (int y = 0; y < gridHeight; ++y)
    {
        for (int x = 0; x < gridWidth; ++x)
        {
            flatGridCurrent.push_back(static_cast<uint8_t>(gridCurrent[y][x]));
            flatGridNext.push_back(static_cast<uint8_t>(gridNext[y][x]));
        }
    }
    // * Copy vectors from host to device
    hipMemcpy(d_gridCurrent, flatGridCurrent.data(), size,
               hipMemcpyHostToDevice);
    hipMemcpy(d_gridNext, flatGridNext.data(), size, hipMemcpyHostToDevice);

    // * Determine the number of blocks per grid.
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // * Start the simulation
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            if (event.type == Event::Closed ||
                Keyboard::isKeyPressed(Keyboard::Escape))
            {
                window.close();
            }
        }
        updateGridKernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_gridCurrent, d_gridNext, gridWidth, gridHeight);

        hipDeviceSynchronize();

        // * We move the memory from GPU to host to render the image
        hipMemcpy(flatGridCurrent.data(), d_gridNext, size,
                   hipMemcpyDeviceToHost);
        // * Start Rendering

        window.clear();

        RectangleShape cell(Vector2f(cellSize, cellSize));

        for (int y = 0; y < gridHeight; ++y)
        {
            for (int x = 0; x < gridWidth; ++x)
            {
                if (flatGridCurrent[y * gridWidth + x])
                {
                    // cell.setPosition(y * cellSize, x * cellSize);
                    cell.setPosition(x * cellSize, y * cellSize);
                    cell.setFillColor(Color::White);
                    window.draw(cell);
                }
            }
        }

        window.display();

        // * We do the memory swap INSIDE the GPU so we do not have to
        // * move the memory from HOST to GPU AGAIN.

        uint8_t* temp = d_gridCurrent;
        d_gridCurrent = d_gridNext;
        d_gridNext = temp;
        // hipMemcpy(d_gridCurrent, flatGridCurrent.data(), size,
        //            hipMemcpyHostToDevice);
    }

    // * Free the GPU Memory when the while loop finishes
    hipFree(d_gridCurrent);
    hipFree(d_gridNext);
}
