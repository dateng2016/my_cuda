#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;
using namespace sf;

// CUDA kernel for updating the grid
__global__ void updateGridKernel(uint8_t* gridCurrent, uint8_t* gridNext,
                                 int gridWidth, int gridHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // x index of cell
    int y = blockIdx.y * blockDim.y + threadIdx.y; // y index of cell

    if (x >= gridWidth || y >= gridHeight)
        return; // Boundary check

    int neighbors = 0;

    // Count neighbors of the current cell
    for (int dx = -1; dx <= 1; dx++)
    {
        for (int dy = -1; dy <= 1; dy++)
        {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && nx < gridWidth && ny >= 0 && ny < gridHeight &&
                !(dx == 0 && dy == 0))
            {
                neighbors += gridCurrent[nx + ny * gridWidth];
            }
        }
    }

    // Conway's Game of Life rules
    int idx = x + y * gridWidth;
    if (gridCurrent[idx])
    {
        gridNext[idx] =
            (neighbors == 2 || neighbors == 3); // Cell remains alive
    }
    else
    {
        gridNext[idx] = (neighbors == 3); // Cell becomes alive
    }
}

void normalMemSimulate(RenderWindow& window, int threadsPerBlock,
                       vector<vector<uint8_t>>& gridCurrent,
                       vector<vector<uint8_t>>& gridNext, int gridWidth,
                       int gridHeight, int cellSize)
{
    uint8_t *d_gridCurrent, *d_gridNext;
    int N = gridWidth * gridHeight;
    size_t size = N * sizeof(uint8_t);
    // * Allocate Memory on GPU
    hipMalloc(&d_gridCurrent, size);
    hipMalloc(&d_gridNext, size);

    // * Flatten the vectors
    vector<uint8_t> flatGridCurrent;
    vector<uint8_t> flatGridNext;
    flatGridCurrent.reserve(gridWidth *
                            gridHeight); // Reserve memory for efficiency
    flatGridNext.reserve(gridWidth *
                         gridHeight); // Reserve memory for efficiency

    for (int y = 0; y < gridHeight; ++y)
    {
        for (int x = 0; x < gridWidth; ++x)
        {
            flatGridCurrent.push_back(static_cast<uint8_t>(gridCurrent[y][x]));
            flatGridNext.push_back(static_cast<uint8_t>(gridNext[y][x]));
        }
    }
    // * Copy vectors from host to device
    hipMemcpy(d_gridCurrent, flatGridCurrent.data(), size,
               hipMemcpyHostToDevice);
    hipMemcpy(d_gridNext, flatGridNext.data(), size, hipMemcpyHostToDevice);

    // * Determine the number of blocks per grid.
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // * Start the simulation
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            if (event.type == Event::Closed ||
                Keyboard::isKeyPressed(Keyboard::Escape))
            {
                window.close();
            }
        }
        cout << blocksPerGrid << endl << threadsPerBlock << endl;
        updateGridKernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_gridCurrent, d_gridNext, gridWidth, gridHeight);

        hipDeviceSynchronize();

        // * We move the memory from GPU to host to render the image
        hipMemcpy(flatGridCurrent.data(), d_gridNext, size,
                   hipMemcpyDeviceToHost);
        // * Start Rendering

        window.clear();

        RectangleShape cell(Vector2f(cellSize, cellSize));

        for (int y = 0; y < gridHeight; ++y)
        {
            for (int x = 0; x < gridWidth; ++x)
            {
                if (flatGridCurrent[y * gridWidth + x])
                {
                    // cell.setPosition(y * cellSize, x * cellSize);
                    cell.setPosition(x * cellSize, y * cellSize);
                    cell.setFillColor(Color::White);
                    window.draw(cell);
                }
            }
        }

        window.display();

        // * We do the memory swap INSIDE the GPU so we do not have to
        // * move the memory from HOST to GPU AGAIN.

        uint8_t* temp = d_gridCurrent;
        d_gridCurrent = d_gridNext;
        d_gridNext = temp;
        // hipMemcpy(d_gridCurrent, flatGridCurrent.data(), size,
        //            hipMemcpyHostToDevice);
    }

    // * Free the GPU Memory when the while loop finishes
    hipFree(d_gridCurrent);
    hipFree(d_gridNext);
}

// void normalMemSimulate2(RenderWindow& window, int threadsPerBlock,
//                        vector<vector<bool>>& gridCurrent,
//                        vector<vector<bool>>& gridNext, int gridWidth,
//                        int gridHeight, int cellSize)
// {

//     // Initialize grid states on host
//     bool *d_gridCurrent, *d_gridNext;

//     // Allocate memory on device (GPU)
//     hipMalloc((void**)&d_gridCurrent, gridWidth * gridHeight *
//     sizeof(bool)); hipMalloc((void**)&d_gridNext, gridWidth * gridHeight *
//     sizeof(bool));

//     // Copy data from host (CPU) to device (GPU)
//     hipMemcpy(d_gridCurrent, gridCurrent.data(),
//                gridWidth * gridHeight * sizeof(bool),
//                hipMemcpyHostToDevice);
//     hipMemcpy(d_gridNext, gridNext.data(),
//                gridWidth * gridHeight * sizeof(bool),
//                hipMemcpyHostToDevice);

//     // Define block size (32 threads per block)
//     dim3 blockDim(threadsPerBlock, 1); // 32 threads in 1D (x-direction)
//     dim3 gridDim((gridWidth + blockDim.x - 1) / blockDim.x,
//                  (gridHeight + blockDim.y - 1) /
//                      blockDim.y); // Grid size to cover all cells

//     // Run the simulation for multiple generations
//     for (int generationCount = 0; window.isOpen(); ++generationCount)
//     {

//         Event event;
//         while (window.pollEvent(event))
//         {
//             if (event.type == Event::Closed ||
//                 Keyboard::isKeyPressed(Keyboard::Escape))
//             {
//                 window.close();
//             }
//         }

//         // Launch CUDA kernel to update the grid
//         updateGridKernel<<<gridDim, blockDim>>>(d_gridCurrent, d_gridNext,
//                                                 gridWidth, gridHeight);

//         // Check for kernel launch errors
//         hipError_t err = hipGetLastError();
//         if (err != hipSuccess)
//         {
//             std::cerr << "CUDA kernel launch failed: "
//                       << hipGetErrorString(err) << std::endl;
//             exit(EXIT_FAILURE);
//         }

//         // Copy the updated grid back to host
//         hipMemcpy(gridCurrent.data(), d_gridNext,
//                    gridWidth * gridHeight * sizeof(bool),
//                    hipMemcpyDeviceToHost);
//         // ! NOTE: the gridCurrent gets changed after the Memcpy happens

//         window.clear();

//         // Draw the grid
//         for (int x = 0; x < gridWidth; ++x)
//         {
//             for (int y = 0; y < gridHeight; ++y)
//             {
//                 if (gridCurrent[x][y])
//                 {
//                     RectangleShape cell(Vector2f(cellSize, cellSize));
//                     cell.setPosition(x * cellSize, y * cellSize);
//                     cell.setFillColor(Color::White);
//                     window.draw(cell);
//                 }
//             }
//         }

//         window.display();

//         // Swap grids for the next generation
//         gridCurrent = gridNext;

//         // Check for performance every 100 generations
//         if (generationCount % 100 == 0)
//         {
//             cout << "Generation " << generationCount << " complete." << endl;
//         }
//     }

//     // Free device memory
//     hipFree(d_gridCurrent);
//     hipFree(d_gridNext);
// }

__global__ void vectorAddKernel(const float* A, const float* B, float* C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

void vectorAdd(const float* A, const float* B, float* C, int N)
{
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate memory on GPU
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "blocksPerGrid = " << blocksPerGrid << std::endl;

    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();
    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i)
    {
        std::cout << "C[" << i << "] = " << C[i] << std::endl;
    }

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
